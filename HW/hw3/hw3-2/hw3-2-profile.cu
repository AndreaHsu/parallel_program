#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//======================
#define DEV_NO 0
hipDeviceProp_t prop;
const int INF = ((1 << 30) - 1);
const int Blocksize = 64;
const int Half = 32;
int* Dist;
int realn, n, m;

int ceil(int a, int b) { return (a + b - 1) / b; }
void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    realn = n;
    n = ceil(n, Blocksize) * Blocksize;
    Dist = (int*) malloc(n * n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < realn; ++i) {
        fwrite(Dist + i * n, sizeof(int), realn, outfile);
    }
    
    fclose(outfile);
}

__global__ void block_FW_p1(int* dist, int round, int n){
    __shared__ int shr[Blocksize][Blocksize];
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row

    int c = round * Blocksize + threadIdx.x;
    int r = round * Blocksize + threadIdx.y;

    shr[y][x] = dist[r * n + c];
    shr[y + Half][x] = dist[(r + Half) * n + c];
    shr[y][x + Half] = dist[r * n + (c + Half)];
    shr[y + Half][x + Half] = dist[(r + Half) * n + (c + Half)];

    __syncthreads();

    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], shr[y][i] + shr[i][x]);
        shr[y + Half][x] = min(shr[y + Half][x], shr[y + Half][i] + shr[i][x]);
        shr[y][x + Half] = min(shr[y][x + Half], shr[y][i] + shr[i][x + Half]);
        shr[y + Half][x + Half] = min(shr[y + Half][x + Half], shr[y + Half][i] + shr[i][x + Half]);
        __syncthreads();
    }
    
    dist[r * n + c] = shr[y][x];
    dist[(r + Half) * n + c] = shr[y + Half][x];
    dist[r * n + (c + Half)] = shr[y][x + Half];
    dist[(r + Half) * n + (c + Half)] = shr[y + Half][x + Half];
    return;
}

__global__ void block_FW_p2(int* dist, int round, int n){
    if(blockIdx.y == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    // I am a thread in the block beside the current pivot block
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row
    // A thread in the current pivot block
    int pivotc = round * Blocksize + threadIdx.x; // col
    int pivotr = round * Blocksize + threadIdx.y; // row
    // The responsible position for me
    int respc = blockIdx.y * Blocksize + threadIdx.x; // if the responsible row is the same as pivotr, calculate the responsible col
    int respr = blockIdx.y * Blocksize + threadIdx.y; // if the responsible col is the same as pivotc, calculate the responsible row
    
    // load pivot
    shr[y][x] = dist[pivotr * n + pivotc];
    shr[y + Half][x] = dist[(pivotr + Half) * n + pivotc];
    shr[y][x + Half] = dist[pivotr * n + (pivotc + Half)];
    shr[y + Half][x + Half] = dist[(pivotr + Half) * n + (pivotc + Half)];

    // load the same row as pivot
    row[y][x] = dist[pivotr * n + respc]; 
    row[y + Half][x] = dist[(pivotr + Half) * n + respc]; 
    row[y][x + Half] = dist[pivotr * n + (respc + Half)]; 
    row[y + Half][x + Half] = dist[(pivotr + Half) * n + (respc + Half)]; 

    // load the same col as pivot
    col[y][x] = dist[respr * n + pivotc];
    col[y + Half][x] = dist[(respr + Half) * n + pivotc];
    col[y][x + Half] = dist[respr * n + (pivotc + Half)];
    col[y + Half][x + Half] = dist[(respr + Half) * n + (pivotc + Half)];

    __syncthreads();

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        row[y][x] = min(row[y][x], shr[y][i] + row[i][x]);
        row[y + Half][x] = min(row[y + Half][x], shr[y + Half][i] + row[i][x]);
        row[y][x + Half] = min(row[y][x + Half], shr[y][i] + row[i][x + Half]);
        row[y + Half][x + Half] = min(row[y + Half][x + Half], shr[y + Half][i] + row[i][x + Half]);

        col[y][x] = min(col[y][x], col[y][i] + shr[i][x]);
        col[y + Half][x] = min(col[y + Half][x], col[y + Half][i] + shr[i][x]);
        col[y][x + Half] = min(col[y][x + Half], col[y][i] + shr[i][x + Half]);
        col[y + Half][x + Half] = min(col[y + Half][x + Half], col[y + Half][i] + shr[i][x + Half]);
        __syncthreads();
    }
    
    dist[pivotr * n + respc] = row[y][x]; 
    dist[(pivotr + Half) * n + respc] = row[y + Half][x]; 
    dist[pivotr * n + (respc + Half)] = row[y][x + Half]; 
    dist[(pivotr + Half) * n + (respc + Half)] = row[y + Half][x + Half]; 

    dist[respr * n + pivotc] = col[y][x];
    dist[(respr + Half) * n + pivotc] = col[y + Half][x];
    dist[respr * n + (pivotc + Half)] = col[y][x + Half];
    dist[(respr + Half) * n + (pivotc + Half)] = col[y + Half][x + Half];
    return;
}

__global__ void block_FW_p3(int* dist, int round, int n){
    if(blockIdx.x == round || blockIdx.y == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    // I am the thread in the block beside the phase2 blocks
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row
    // my real col and real row in the whole matrix
    int realc = blockIdx.x * Blocksize + threadIdx.x;
    int realr = blockIdx.y * Blocksize + threadIdx.y;
    // The needed position to calculate my value
    int neededc = round * Blocksize + threadIdx.x; // if the needed row is the same as me, calculate the needed col
    int neededr = round * Blocksize + threadIdx.y; // if the needed col is the same as me, calculate the needed col

    // load the same row as me
    row[y][x] = dist[realr * n + neededc]; 
    row[y + Half][x] = dist[(realr + Half) * n + neededc]; 
    row[y][x + Half] = dist[realr * n + (neededc + Half)]; 
    row[y + Half][x + Half] = dist[(realr + Half) * n + (neededc + Half)]; 

    // load the same column as me
    col[y][x] = dist[neededr * n + realc];
    col[y + Half][x] = dist[(neededr + Half) * n + realc];
    col[y][x + Half] = dist[neededr * n + (realc + Half)];
    col[y + Half][x + Half] = dist[(neededr + Half) * n + (realc + Half)];
    __syncthreads();

    shr[y][x] = dist[realr * n + realc];
    shr[y + Half][x] = dist[(realr + Half) * n + realc];
    shr[y][x + Half] = dist[realr * n + (realc + Half)];
    shr[y + Half][x + Half] = dist[(realr + Half) * n + (realc + Half)];

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], row[y][i] + col[i][x]);
        shr[y + Half][x] = min(shr[y + Half][x], row[y + Half][i] + col[i][x]);
        shr[y][x + Half] = min(shr[y][x + Half], row[y][i] + col[i][x + Half]);
        shr[y + Half][x + Half] = min(shr[y + Half][x + Half], row[y + Half][i] + col[i][x + Half]);
    }
    
    dist[realr * n + realc] = shr[y][x];
    dist[(realr + Half) * n + realc] = shr[y + Half][x];
    dist[realr * n + (realc + Half)] = shr[y][x + Half];
    dist[(realr + Half) * n + (realc + Half)] = shr[y + Half][x + Half];
    return;
}


int main(int argc, char* argv[]) {
    struct timespec io_instart, io_inend, io_outstart, io_outend;
    double io_elapsed = 0;
    clock_gettime(CLOCK_MONOTONIC, &io_instart);
    input(argv[1]);
    clock_gettime(CLOCK_MONOTONIC, &io_inend);
    io_elapsed += (io_inend.tv_sec - io_instart.tv_sec) + (io_inend.tv_nsec - io_instart.tv_nsec) / 1e9;
    int* ddist;
    // hipHostRegister(Dist, n * n * sizeof(int), hipHostRegisterDefault);
    hipMalloc(&ddist, n * n * sizeof(int));
    hipMemcpy(ddist, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

    // hipGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    // maxThreasPerBlock = 1024, sharedMemPerBlock = 49152
    int B = n / Blocksize;
    dim3 num_blocks_p1(1, 1);
    dim3 num_blocks_p2(1, B);
    dim3 num_blocks_p3(B, B);
    dim3 num_threads(Half, Half);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i = 0; i < B; i++){
        block_FW_p1<<<num_blocks_p1, num_threads>>>(ddist, i, n);
        block_FW_p2<<<num_blocks_p2, num_threads>>>(ddist, i, n);
        block_FW_p3<<<num_blocks_p3, num_threads>>>(ddist, i, n);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %f\n", elapsedTime);

    hipMemcpy(Dist, ddist, n * n * sizeof(int), hipMemcpyDeviceToHost);
    
    clock_gettime(CLOCK_MONOTONIC, &io_outstart);
    output(argv[2]);
    clock_gettime(CLOCK_MONOTONIC, &io_outend);
    io_elapsed += (io_outend.tv_sec - io_outstart.tv_sec) + (io_outend.tv_nsec - io_outstart.tv_nsec) / 1e9;
    printf("IO Elapsed Time: %f\n", io_elapsed);
    return 0;
}