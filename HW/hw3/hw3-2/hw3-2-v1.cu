#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//======================
#define DEV_NO 0
hipDeviceProp_t prop;
const int INF = ((1 << 30) - 1);
const int Blocksize = 32;
int* Dist;
int realn, n, m;

int ceil(int a, int b) { return (a + b - 1) / b; }
void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    realn = n;
    n = ceil(n, Blocksize) * Blocksize;
    Dist = (int*) malloc(n * n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < realn; ++i) {
        fwrite(Dist + i * n, sizeof(int), realn, outfile);
    }
    
    fclose(outfile);
}

__global__ void block_FW_p1(int* dist, int round, int n){
    __shared__ int shr[Blocksize][Blocksize];
    int x = threadIdx.x;
    int y = threadIdx.y;

    int realx = threadIdx.x + round * Blocksize;
    int realy = threadIdx.y + round * Blocksize;

    shr[y][x] = dist[realx * n + realy];

    __syncthreads();

    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], shr[y][i] + shr[i][x]);
        __syncthreads();
    }
    
    dist[realx * n + realy] = shr[y][x];
    return;
}

__global__ void block_FW_p2(int* dist, int round, int n){
    if(blockIdx.y == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    int x = threadIdx.x;
    int y = threadIdx.y;

    int realx = round * Blocksize + threadIdx.x;
    int realy = round * Blocksize + threadIdx.y;
    int realSameRowPos = realx * n + (blockIdx.y * Blocksize + threadIdx.y);
    int realSameColPos = (blockIdx.y * Blocksize + threadIdx.x) * n + realy;
    
    shr[y][x] = dist[realx * n + realy];
    row[y][x] = dist[realSameRowPos]; 
    col[y][x] = dist[realSameColPos];

    __syncthreads();

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        row[y][x] = min(row[y][x], row[y][i] + shr[i][x]);
        col[y][x] = min(col[y][x], shr[y][i] + col[i][x]);
    }
    
    dist[realSameRowPos] = row[y][x];
    dist[realSameColPos] = col[y][x];
    return;
}

__global__ void block_FW_p3(int* dist, int round, int n){
    if(blockIdx.x == round || blockIdx.y == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    int x = threadIdx.x;
    int y = threadIdx.y;

    int realx = blockIdx.x * Blocksize + threadIdx.x;
    int realy = blockIdx.y * Blocksize + threadIdx.y;
    // if(realx >= n || realy >= n) return;
    int realSameRowPos = (round * Blocksize + threadIdx.x) * n + realy;
    int realSameColPos = realx * n + (round * Blocksize + threadIdx.y);
    
    shr[y][x] = dist[realx * n + realy];
    row[y][x] = dist[realSameRowPos]; 
    col[y][x] = dist[realSameColPos];

    __syncthreads();

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], row[y][i] + col[i][x]);
    }
    
    dist[realx * n + realy] = shr[y][x];
    return;
}


int main(int argc, char* argv[]) {
    input(argv[1]);
    int* ddist;
    hipMalloc(&ddist, n * n * sizeof(int));
    hipMemcpy(ddist, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

    // cudaGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    // maxThreasPerBlock = 1024, sharedMemPerBlock = 49152
    int B = n / Blocksize;
    dim3 num_blocks_p1(1, 1);
    dim3 num_blocks_p2(1, B);
    dim3 num_blocks_p3(B, B);
    dim3 num_threads(32, 32);

    for(int i = 0; i < B; i++){
        block_FW_p1<<<num_blocks_p1, num_threads>>>(ddist, i, n);
        block_FW_p2<<<num_blocks_p2, num_threads>>>(ddist, i, n);
        block_FW_p3<<<num_blocks_p3, num_threads>>>(ddist, i, n);
    }

    hipMemcpy(Dist, ddist, n * n * sizeof(int), hipMemcpyDeviceToHost);
    output(argv[2]);
    return 0;
}