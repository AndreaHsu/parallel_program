#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define DEV_NO 0
hipDeviceProp_t prop;
const int INF = ((1 << 30) - 1);
const int Blocksize = 64;
const int Half = 32;
int* Dist;
int realn, n, m;

int ceil(int a, int b) { return (a + b - 1) / b; }
void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    realn = n;
    n = ceil(n, Blocksize) * Blocksize;
    Dist = (int*) malloc(n * n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < realn; ++i) {
        fwrite(Dist + i * n, sizeof(int), realn, outfile);
    }
    
    fclose(outfile);
}

__global__ void block_FW_p1(int* dist, int round, int n){
    __shared__ int shr[Blocksize][Blocksize];
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row

    int c = round * Blocksize + threadIdx.x;
    int r = round * Blocksize + threadIdx.y;

    shr[y][x] = dist[r * n + c];
    shr[y + Half][x] = dist[(r + Half) * n + c];
    shr[y][x + Half] = dist[r * n + (c + Half)];
    shr[y + Half][x + Half] = dist[(r + Half) * n + (c + Half)];

    __syncthreads();

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], shr[y][i] + shr[i][x]);
        shr[y + Half][x] = min(shr[y + Half][x], shr[y + Half][i] + shr[i][x]);
        shr[y][x + Half] = min(shr[y][x + Half], shr[y][i] + shr[i][x + Half]);
        shr[y + Half][x + Half] = min(shr[y + Half][x + Half], shr[y + Half][i] + shr[i][x + Half]);
        __syncthreads();
    }
    
    dist[r * n + c] = shr[y][x];
    dist[(r + Half) * n + c] = shr[y + Half][x];
    dist[r * n + (c + Half)] = shr[y][x + Half];
    dist[(r + Half) * n + (c + Half)] = shr[y + Half][x + Half];
    return;
}

__global__ void block_FW_p2(int* dist, int round, int n){
    if(blockIdx.y == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    // I am a thread in the block beside the current pivot block
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row
    // A thread in the current pivot block
    int pivotc = round * Blocksize + threadIdx.x; // col
    int pivotr = round * Blocksize + threadIdx.y; // row
    // The responsible position for me
    int respc = blockIdx.y * Blocksize + threadIdx.x; // if the responsible row is the same as pivotr, calculate the responsible col
    int respr = blockIdx.y * Blocksize + threadIdx.y; // if the responsible col is the same as pivotc, calculate the responsible row
    
    // load pivot
    shr[y][x] = dist[pivotr * n + pivotc];
    shr[y + Half][x] = dist[(pivotr + Half) * n + pivotc];
    shr[y][x + Half] = dist[pivotr * n + (pivotc + Half)];
    shr[y + Half][x + Half] = dist[(pivotr + Half) * n + (pivotc + Half)];

    // load the same row as pivot
    row[y][x] = dist[pivotr * n + respc]; 
    row[y + Half][x] = dist[(pivotr + Half) * n + respc]; 
    row[y][x + Half] = dist[pivotr * n + (respc + Half)]; 
    row[y + Half][x + Half] = dist[(pivotr + Half) * n + (respc + Half)]; 

    // load the same col as pivot
    col[y][x] = dist[respr * n + pivotc];
    col[y + Half][x] = dist[(respr + Half) * n + pivotc];
    col[y][x + Half] = dist[respr * n + (pivotc + Half)];
    col[y + Half][x + Half] = dist[(respr + Half) * n + (pivotc + Half)];

    __syncthreads();

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        row[y][x] = min(row[y][x], shr[y][i] + row[i][x]);
        row[y + Half][x] = min(row[y + Half][x], shr[y + Half][i] + row[i][x]);
        row[y][x + Half] = min(row[y][x + Half], shr[y][i] + row[i][x + Half]);
        row[y + Half][x + Half] = min(row[y + Half][x + Half], shr[y + Half][i] + row[i][x + Half]);

        col[y][x] = min(col[y][x], col[y][i] + shr[i][x]);
        col[y + Half][x] = min(col[y + Half][x], col[y + Half][i] + shr[i][x]);
        col[y][x + Half] = min(col[y][x + Half], col[y][i] + shr[i][x + Half]);
        col[y + Half][x + Half] = min(col[y + Half][x + Half], col[y + Half][i] + shr[i][x + Half]);
        __syncthreads();
    }
    
    dist[pivotr * n + respc] = row[y][x]; 
    dist[(pivotr + Half) * n + respc] = row[y + Half][x]; 
    dist[pivotr * n + (respc + Half)] = row[y][x + Half]; 
    dist[(pivotr + Half) * n + (respc + Half)] = row[y + Half][x + Half]; 

    dist[respr * n + pivotc] = col[y][x];
    dist[(respr + Half) * n + pivotc] = col[y + Half][x];
    dist[respr * n + (pivotc + Half)] = col[y][x + Half];
    dist[(respr + Half) * n + (pivotc + Half)] = col[y + Half][x + Half];
    return;
}

__global__ void block_FW_p3(int* dist, int round, int n, int row_offset){
    if(blockIdx.x == round || (blockIdx.y + row_offset) == round) return;
    __shared__ int shr[Blocksize][Blocksize];
    __shared__ int row[Blocksize][Blocksize];
    __shared__ int col[Blocksize][Blocksize];
    // I am the thread in the block beside the phase2 blocks
    int x = threadIdx.x; // col
    int y = threadIdx.y; // row
    // my real col and real row in the whole matrix
    int realc = blockIdx.x * Blocksize + threadIdx.x;
    int realr = (blockIdx.y + row_offset) * Blocksize + threadIdx.y;
    // The needed position to calculate my value
    int neededc = round * Blocksize + threadIdx.x; // if the needed row is the same as me, calculate the needed col
    int neededr = round * Blocksize + threadIdx.y; // if the needed col is the same as me, calculate the needed col

    // load the same row as me
    row[y][x] = dist[realr * n + neededc]; 
    row[y + Half][x] = dist[(realr + Half) * n + neededc]; 
    row[y][x + Half] = dist[realr * n + (neededc + Half)]; 
    row[y + Half][x + Half] = dist[(realr + Half) * n + (neededc + Half)]; 

    // load the same column as me
    col[y][x] = dist[neededr * n + realc];
    col[y + Half][x] = dist[(neededr + Half) * n + realc];
    col[y][x + Half] = dist[neededr * n + (realc + Half)];
    col[y + Half][x + Half] = dist[(neededr + Half) * n + (realc + Half)];
    __syncthreads();

    shr[y][x] = dist[realr * n + realc];
    shr[y + Half][x] = dist[(realr + Half) * n + realc];
    shr[y][x + Half] = dist[realr * n + (realc + Half)];
    shr[y + Half][x + Half] = dist[(realr + Half) * n + (realc + Half)];

    #pragma unroll 32
    for(int i = 0; i < Blocksize; i++){
        shr[y][x] = min(shr[y][x], row[y][i] + col[i][x]);
        shr[y + Half][x] = min(shr[y + Half][x], row[y + Half][i] + col[i][x]);
        shr[y][x + Half] = min(shr[y][x + Half], row[y][i] + col[i][x + Half]);
        shr[y + Half][x + Half] = min(shr[y + Half][x + Half], row[y + Half][i] + col[i][x + Half]);
    }
    
    dist[realr * n + realc] = shr[y][x];
    dist[(realr + Half) * n + realc] = shr[y + Half][x];
    dist[realr * n + (realc + Half)] = shr[y][x + Half];
    dist[(realr + Half) * n + (realc + Half)] = shr[y + Half][x + Half];
    return;
}


int main(int argc, char* argv[]) { 
    input(argv[1]);
    int* ddist[2];
    // cudaHostRegister(Dist, n * n * sizeof(int), cudaHostRegisterDefault);

    // cudaGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    // maxThreasPerBlock = 1024, sharedMemPerBlock = 49152
    int B = n / Blocksize;
    dim3 num_blocks_p1(1, 1);
    dim3 num_blocks_p2(1, B);
    dim3 num_threads(32, 32);
    #pragma omp parallel num_threads(2)
    {
        int id = omp_get_thread_num();
        hipSetDevice(id);
        hipDeviceEnablePeerAccess(!id, 0);
        hipMalloc(&(ddist[id]), n * n * sizeof(int));
        hipMemcpy(ddist[id], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
        
        dim3 num_blocks_p3(B, B / 2);
        int row_offset = 0;
        if(id){
            row_offset = B / 2;
            if(B & 1) num_blocks_p3.y++;
        } 

        for(int i = 0; i < B; i++){
            if(!id && i < B / 2){
                hipMemcpyPeer(ddist[1] + i * Blocksize * n, 1, ddist[0] + i * Blocksize * n, 0, Blocksize * n * sizeof(int));
            }else if(id && i >= B / 2){
                hipMemcpyPeer(ddist[0] + i * Blocksize * n, 0, ddist[1] + i * Blocksize * n, 1, Blocksize * n * sizeof(int));
            }
            #pragma omp barrier
            block_FW_p1<<<num_blocks_p1, num_threads>>>(ddist[id], i, n);
            block_FW_p2<<<num_blocks_p2, num_threads>>>(ddist[id], i, n);
            block_FW_p3<<<num_blocks_p3, num_threads>>>(ddist[id], i, n, row_offset);
        }
        hipMemcpy(Dist + row_offset * Blocksize * n, ddist[id] + row_offset * Blocksize * n, num_blocks_p3.y * Blocksize * n * sizeof(int), hipMemcpyDeviceToHost);
    }

    output(argv[2]);
    return 0;
}